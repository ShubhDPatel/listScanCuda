#include "hip/hip_runtime.h"
/* ACADEMIC INTEGRITY PLEDGE                                              */
/*                                                                        */
/* - I have not used source code obtained from another student nor        */
/*   any other unauthorized source, either modified or unmodified.        */
/*                                                                        */
/* - All source code and documentation used in my program is either       */
/*   my original work or was derived by me from the source code           */
/*   published in the textbook for this course or presented in            */
/*   class.                                                               */
/*                                                                        */
/* - I have not discussed coding details about this project with          */
/*   anyone other than my instructor. I understand that I may discuss     */
/*   the concepts of this program with other students and that another    */
/*   student may help me debug my program so long as neither of us        */
/*   writes anything during the discussion or modifies any computer       */
/*   file during the discussion.                                          */
/*                                                                        */
/* - I have violated neither the spirit nor letter of these restrictions. */
/*                                                                        */
/*                                                                        */
/*                                                                        */
/* Signed: Shubh Patel  Date: 4/6/2024                                    */
/*                                                                        */
/*                                                                        */
/* CPSC 677 CUDA Prefix Sum lab, Version 1.02, Spring 2024.               */

#include "helper_timer.h"
#include <stdio.h>
#include <stdlib.h>

// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// + lst[n-1]}

#define BLOCK_SIZE 512 //@@ You can change this

__global__ void scan(int* input, int* output, int* aux, int len)
{
    //@@ Modify the body of this function to complete the functionality of
    //@@ the scan on the device
    //@@ You may need multiple kernel calls; write your kernels before this
    //@@ function and call them from here

    // Initialize the shared memory
    extern __shared__ float temp[];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len)
    {
        temp[2 * threadIdx.x] = input[i];
        // temp[2 * threadIdx.x + 1] = input[i];
    }

    for (unsigned int stride = 1; stride <= BLOCK_SIZE; stride <<= 1)
    {
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index < 2 * BLOCK_SIZE)
        {
            temp[index] += temp[index - stride];
        }
        __syncthreads();
    }

    for (unsigned int stride = BLOCK_SIZE / 2; stride > 0; stride >>= 1)
    {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < 2 * BLOCK_SIZE)
        {
            temp[index + stride] += temp[index];
        }
    }
    __syncthreads();
    if (i < len)
    {
        output[i] = temp[2 * threadIdx.x];
        aux[i] = temp[2 * threadIdx.x + 1];
    }
}

int main(int argc, char** argv)
{
    int* hostInput; // The input 1D list
    int* hostOutput; // The output list
    int* expectedOutput;
    int* deviceInput;
    int* deviceOutput;
    int *deviceAuxArray, *deviceAuxScannedArray;
    int numElements; // number of elements in the list

    FILE *infile, *outfile;
    int inputLength, outputLength;
    StopWatchLinux stw;
    unsigned int blog = 1;

    // Import host input data
    stw.start();
    if ((infile = fopen("input.raw", "r")) == NULL)
    {
        printf("Cannot open input.raw.\n");
        exit(EXIT_FAILURE);
    }
    fscanf(infile, "%i", &inputLength);
    hostInput = (int*)malloc(sizeof(int) * inputLength);
    for (int i = 0; i < inputLength; i++)
        fscanf(infile, "%i", &hostInput[i]);
    fclose(infile);
    numElements = inputLength;
    hostOutput = (int*)malloc(numElements * sizeof(int));
    stw.stop();
    printf("Importing data and creating memory on host: %f ms\n", stw.getTime());

    if (blog)
        printf("*** The number of input elements in the input is %i\n", numElements);

    stw.reset();
    stw.start();

    hipMalloc((void**)&deviceInput, numElements * sizeof(int));
    hipMalloc((void**)&deviceOutput, numElements * sizeof(int));

    hipMalloc(&deviceAuxArray, (BLOCK_SIZE << 1) * sizeof(int));
    hipMalloc(&deviceAuxScannedArray, (BLOCK_SIZE << 1) * sizeof(int));

    stw.stop();
    printf("Allocating GPU memory: %f ms\n", stw.getTime());

    stw.reset();
    stw.start();

    hipMemset(deviceOutput, 0, numElements * sizeof(int));

    stw.stop();
    printf("Clearing output memory: %f ms\n", stw.getTime());

    stw.reset();
    stw.start();

    hipMemcpy(deviceInput, hostInput, numElements * sizeof(int),
        hipMemcpyHostToDevice);

    stw.stop();
    printf("Copying input memory to the GPU: %f ms\n", stw.getTime());

    //@@ Initialize the grid and block dimensions here
    dim3 blockDim(BLOCK_SIZE, 1, 1);
    dim3 gridDim(ceil((float)numElements / blockDim.x), 1, 1);

    stw.reset();
    stw.start();

    //@@ Modify this to complete the functionality of the scan
    //@@ on the device
    scan<<<gridDim, blockDim, 2 * BLOCK_SIZE * sizeof(float)>>>(deviceInput, deviceOutput, deviceAuxArray, BLOCK_SIZE);

    hipDeviceSynchronize();

    stw.stop();
    printf("Performing CUDA computation: %f ms\n", stw.getTime());

    stw.reset();
    stw.start();

    hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(int),
        hipMemcpyDeviceToHost);

    stw.stop();
    printf("Copying output memory to the CPU: %f ms\n", stw.getTime());

    stw.reset();
    stw.start();

    hipFree(deviceInput);
    hipFree(deviceOutput);
    hipFree(deviceAuxArray);
    hipFree(deviceAuxScannedArray);

    stw.stop();
    printf("Freeing GPU Memory: %f ms\n", stw.getTime());

    if ((outfile = fopen("output.raw", "r")) == NULL)
    {
        printf("Cannot open output.raw.\n");
        exit(EXIT_FAILURE);
    }
    fscanf(outfile, "%i", &outputLength);
    expectedOutput = (int*)malloc(sizeof(int) * outputLength);
    for (int i = 0; i < outputLength; i++)
        fscanf(outfile, "%i", &expectedOutput[i]);
    fclose(outfile);

    int test = 1;
    for (int i = 0; i < outputLength; i++)
    {
        if (expectedOutput[i] != hostOutput[i])
            printf("%i %i %i\n", i, expectedOutput[i], hostOutput[i]);
        test = test && (expectedOutput[i] == hostOutput[i]);
    }

    if (test)
        printf("Results correct.\n");
    else
        printf("Results incorrect.\n");

    free(hostInput);
    hipHostFree(hostOutput);
    free(expectedOutput);

    return 0;
}
